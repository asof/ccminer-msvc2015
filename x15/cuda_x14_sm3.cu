#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#include "cuda_helper.h"

#define SM3_DIGEST_LENGTH	32
#define SM3_BLOCK_SIZE		64
#define SM3_CBLOCK		(SM3_BLOCK_SIZE)
#define SM3_HMAC_SIZE		(SM3_DIGEST_LENGTH)


#define cpu_to_be16(v) (((v)<< 8) | ((v)>>8))
#define cpu_to_be32(v) (((v)>>24) | (((v)>>8)&0xff00) | (((v)<<8)&0xff0000) | ((v)<<24))
#define be16_to_cpu(v) cpu_to_be16(v)
#define be32_to_cpu(v) cpu_to_be32(v)

//ROTATELEFT(X,n) = (((X)<<(n)) | ((X)>>(32-(n))))
#define ROTATELEFT(x, bits) __funnelshift_l(x, x, bits)

#define P0(x) ((x) ^  ROTATELEFT((x),9)  ^ ROTATELEFT((x),17))
#define P1(x) ((x) ^  ROTATELEFT((x),15) ^ ROTATELEFT((x),23))

#define FF0(x,y,z) ( (x) ^ (y) ^ (z))
#define FF1(x,y,z) (((x) & (y)) | ( (x) & (z)) | ( (y) & (z)))

#define GG0(x,y,z) ( (x) ^ (y) ^ (z))
#define GG1(x,y,z) (((x) & (y)) | ( (~(x)) & (z)) )


__device__
void sm3_compress_gpu(uint32_t digest[8], const uint32_t pblock[64])
{
	int j;
	uint32_t W[68], W1[64];
	//const uint32_t *pblock = (const uint32_t *)block;

	uint32_t A = digest[0];
	uint32_t B = digest[1];
	uint32_t C = digest[2];
	uint32_t D = digest[3];
	uint32_t E = digest[4];
	uint32_t F = digest[5];
	uint32_t G = digest[6];
	uint32_t H = digest[7];
	uint32_t SS1, SS2, TT1, TT2, T[64];

	for (j = 0; j < 16; j++) {
		W[j] = cpu_to_be32(pblock[j]);
	}
	for (j = 16; j < 68; j++) {
		W[j] = P1(W[j - 16] ^ W[j - 9] ^ ROTATELEFT(W[j - 3], 15)) ^ ROTATELEFT(W[j - 13], 7) ^ W[j - 6];;
	}
	for (j = 0; j < 64; j++) {
		W1[j] = W[j] ^ W[j + 4];
	}

	for (j = 0; j < 16; j++) {

		T[j] = 0x79CC4519;
		SS1 = ROTATELEFT((ROTATELEFT(A, 12) + E + ROTATELEFT(T[j], j)), 7);
		SS2 = SS1 ^ ROTATELEFT(A, 12);
		TT1 = FF0(A, B, C) + D + SS2 + W1[j];
		TT2 = GG0(E, F, G) + H + SS1 + W[j];
		D = C;
		C = ROTATELEFT(B, 9);
		B = A;
		A = TT1;
		H = G;
		G = ROTATELEFT(F, 19);
		F = E;
		E = P0(TT2);
	}

	for (j = 16; j < 64; j++) {

		T[j] = 0x7A879D8A;
		SS1 = ROTATELEFT((ROTATELEFT(A, 12) + E + ROTATELEFT(T[j], j)), 7);
		SS2 = SS1 ^ ROTATELEFT(A, 12);
		TT1 = FF1(A, B, C) + D + SS2 + W1[j];
		TT2 = GG1(E, F, G) + H + SS1 + W[j];
		D = C;
		C = ROTATELEFT(B, 9);
		B = A;
		A = TT1;
		H = G;
		G = ROTATELEFT(F, 19);
		F = E;
		E = P0(TT2);
	}

	digest[0] ^= A;
	digest[1] ^= B;
	digest[2] ^= C;
	digest[3] ^= D;
	digest[4] ^= E;
	digest[5] ^= F;
	digest[6] ^= G;
	digest[7] ^= H;
}


__global__ void x14_sm3_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint32_t *g_hash)
{
    uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
		uint32_t digest[8];
		unsigned char block[64]; 
		//sm3_ctx_t ctxData;
		//sm3_ctx_t *ctx = &ctxData;
		uint32_t nounce = (startNounce + thread);

		uint32_t hashPosition = nounce - startNounce;
		uint32_t* data = &g_hash[hashPosition * 16];

		memset(digest, 0, 32);
		memset(block, 0, 64);

		digest[0] = 0x7380166F;
		digest[1] = 0x4914B2B9;
		digest[2] = 0x172442D7;
		digest[3] = 0xDA8A0600;
		digest[4] = 0xA96F30BC;
		digest[5] = 0x163138AA;
		digest[6] = 0xE38DEE4D;
		digest[7] = 0xB0FB0E4E;

		sm3_compress_gpu(digest, data);

		uint32_t *pdigest = (uint32_t *)data;
		uint32_t *count = (uint32_t *)(block + SM3_BLOCK_SIZE - 8);

		block[0] = 0x80;
		memset(block + 1, 0, SM3_BLOCK_SIZE - 9);

		count[0] = 0;
		count[1] = cpu_to_be32((uint32_t)1 << 9);

		sm3_compress_gpu(digest, (uint32_t *)block);
		for (int i = 0; i < 8; i++)
			pdigest[i] = cpu_to_be32(digest[i]);
		memset(&pdigest[8], 0, 32);
		//memset(digest, 0, 32);
		//memset(block, 0, 64);
		//if (thread == 0)
		//	printf("%X %X %X %X\n", pdigest[0], pdigest[1], pdigest[2], pdigest[3]);
    }
}


__host__ 
void x14_sm3_cpu_init(int thr_id, uint32_t threads)
{
}

__host__
void x14_sm3_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    const uint32_t threadsperblock = 64;

    dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	x14_sm3_gpu_hash_64 << <grid, block >> >(threads, startNounce, d_hash);
	MyStreamSynchronize(NULL, order, thr_id);
}
